#include "hip/hip_runtime.h"
#include "../../../devices/cuda/cuda_handle.h"
#include "../../utils.h"
#include "../blas.h"
#include "matmul_cuda.h"
#include <hipblas.h>
#include <hip/hip_fp16.h>

template<typename Tdata>
infiniopStatus_t matmul_cuda(MatmulCudaDescriptor_t desc, void *c, float beta, void const *a, void const *b, float alpha, void *stream) {
    auto info = desc->info;

    if (info.is_transed) {
        std::swap(a, b);
    }

    Tdata alpha_, beta_;
    hipDataType a_type, b_type, c_type;
    hipblasComputeType_t compute_type;

    if constexpr (std::is_same<Tdata, half>::value) {
        alpha_ = __float2half(alpha);
        beta_ = __float2half(beta);
        a_type = b_type = c_type = HIP_R_16F;
        compute_type = HIPBLAS_COMPUTE_16F;
    } else {
        alpha_ = alpha;
        beta_ = beta;
        a_type = b_type = c_type = HIP_R_32F;
        compute_type = HIPBLAS_COMPUTE_32F_FAST_TF32;
    }

    auto op_a = info.a_matrix.row_stride == 1 ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    auto op_b = info.b_matrix.row_stride == 1 ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    use_cublas(desc->cublas_handles_t, desc->device_id, (hipStream_t) stream,
               [&](hipblasHandle_t handle) { hipblasGemmStridedBatchedEx(
                                                handle,
                                                op_a,
                                                op_b,
                                                info.m,
                                                info.n,
                                                info.k,
                                                &alpha_,
                                                a,
                                                a_type,
                                                info.a_matrix.ld(),
                                                info.a_matrix.stride,
                                                b,
                                                b_type,
                                                info.b_matrix.ld(),
                                                info.b_matrix.stride,
                                                &beta_,
                                                c,
                                                c_type,
                                                info.c_matrix.ld(),
                                                info.c_matrix.stride,
                                                info.batch,
                                                compute_type,
                                                CUBLAS_GEMM_DEFAULT_TENSOR_OP); });
    return STATUS_SUCCESS;
}

infiniopStatus_t cudaMatmul(MatmulCudaDescriptor_t desc,
                            void *workspace,
                            uint64_t workspace_size,
                            void *c,
                            void const *a,
                            void const *b,
                            void *stream) {
    if (desc->dtype == F16) {
        return matmul_cuda<half>(desc, c, desc->beta, a, b, desc->alpha, stream);
    }
    if (desc->dtype == F32) {
        return matmul_cuda<float>(desc, c, desc->beta, a, b, desc->alpha, stream);
    }
    return STATUS_BAD_TENSOR_DTYPE;
}
